#include "hip/hip_runtime.h"
#include "gelu_cuda.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void gelu_cuda(const float* input, float* output, int s) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    constexpr float y = 0.797885f; // sqrt(2.0 / M_PI)
    constexpr float w = 0.0356774f; // y * 0.044715
    if (idx < s) {
        output[idx] = input[idx] * (1.0f / (1.0f + __expf(-2.0f * (input[idx] * (y + w * input[idx] * input[idx])))));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    int s = input.size();
    std::vector<float> output(s);
    if (s == 0) return output;

    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, 0);

    int length = s * sizeof(float);

    float *d_input, *d_output;
    hipMalloc(&d_input, length);
    hipMalloc(&d_output, length);

    hipMemcpy(d_input, input.data(), length, hipMemcpyHostToDevice);

    int blockSize = dev_prop.maxThreadsPerBlock;
    int gridSize = (s + blockSize - 1) / blockSize;

    gelu_cuda<<<gridSize, blockSize>>>(d_input, d_output, s);

    hipMemcpy(output.data(), d_output, length, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}

#include "hip/hip_runtime.h"
// Copyright (c) 2024 Zakharov Artem
#include "gelu_cuda.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void gelu_kernel(const float* input, float* output, size_t n) {
    constexpr float SQRT_TWO_OVER_PI = 0.797885;
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind < n) {
        float x = input[ind];
        output[ind] = 0.5f * x * (1 + static_cast<float>(tanh(SQRT_TWO_OVER_PI * x * (1 + 0.044715f * x * x))));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, 0);
    size_t n = input.size();
    size_t bytes_size = n * sizeof(float);
    const size_t threads_per_block = dev_prop.maxThreadsPerBlock;
    const size_t num_blocks = (n + threads_per_block - 1) / threads_per_block;
    std::vector<float> result(n);

    float *input_dev = nullptr;
    float *output_dev = nullptr;

    hipMalloc(reinterpret_cast<void**>(&input_dev), bytes_size);
    hipMalloc(reinterpret_cast<void**>(&output_dev), bytes_size);
    hipMemcpy(reinterpret_cast<void*>(input_dev),
               reinterpret_cast<const void*>(input.data()),
               bytes_size, hipMemcpyHostToDevice);

    gelu_kernel<<<num_blocks, threads_per_block>>>(input_dev, output_dev, n);
    hipMemcpy(reinterpret_cast<void*>(result.data()),
               reinterpret_cast<const void*>(output_dev),
               bytes_size, hipMemcpyDeviceToHost);

    hipFree(reinterpret_cast<void*>(input_dev));
    hipFree(reinterpret_cast<void*>(output_dev));
    return result;
}

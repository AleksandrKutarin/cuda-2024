
#include <hip/hip_runtime.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <stdexcept>

__global__ void normalize_kernel(float* data, size_t size, float norm_factor) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] *= norm_factor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {

    int n = input.size() / (2 * batch);

    hipfftComplex* d_input;
    hipfftComplex* d_output;
    size_t size = input.size() * sizeof(float) / 2;

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, input.data(), size, hipMemcpyHostToDevice);

    hipfftHandle plan;
    if (hipfftPlan1d(&plan, n, HIPFFT_C2C, batch) != HIPFFT_SUCCESS) {
        hipFree(d_input);
        hipFree(d_output);
        throw std::runtime_error("Failed to create cuFFT plan.");
    }

    if (hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        hipfftDestroy(plan);
        hipFree(d_input);
        hipFree(d_output);
        throw std::runtime_error("Failed to execute forward FFT.");
    }

    if (hipfftExecC2C(plan, d_output, d_input, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        hipfftDestroy(plan);
        hipFree(d_input);
        hipFree(d_output);
        throw std::runtime_error("Failed to execute inverse FFT.");
    }

    int threadsPerBlock = 256;
    int numBlocks = (input.size() + threadsPerBlock - 1) / threadsPerBlock;
    normalize_kernel<<<numBlocks, threadsPerBlock>>>(reinterpret_cast<float*>(d_input), input.size(), 1.0f / n);

    hipDeviceSynchronize();

    std::vector<float> output(input.size());
    hipMemcpy(output.data(), d_input, size, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_input);
    hipFree(d_output);

    return output;
}

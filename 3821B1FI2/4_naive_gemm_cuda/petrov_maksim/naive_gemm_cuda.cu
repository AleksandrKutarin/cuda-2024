#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "naive_gemm_cuda.h"

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

const int TILE_SIZE = 32;

__global__ void gemm_kernel(const float* a, const float* b, float* c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x; 

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a, const std::vector<float>& b, int n) {
    size_t size = n * n * sizeof(float);

    float* d_a;
    float* d_b;
    float* d_c;

    checkCudaError(hipMalloc((void**)&d_a, size), "Failed to allocate memory for d_a");
    checkCudaError(hipMalloc((void**)&d_b, size), "Failed to allocate memory for d_b");
    checkCudaError(hipMalloc((void**)&d_c, size), "Failed to allocate memory for d_c");

    checkCudaError(hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice), "Failed to copy data for d_a");
    checkCudaError(hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice), "Failed to copy data for d_b");

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((n + TILE_SIZE - 1) / TILE_SIZE, (n + TILE_SIZE - 1) / TILE_SIZE);

    gemm_kernel << <numBlocks, threadsPerBlock >> > (d_a, d_b, d_c, n);
    checkCudaError(hipGetLastError(), "Error when starting the kernel");

    checkCudaError(hipDeviceSynchronize(), "Synchronization error");

    std::vector<float> c(n * n);
    checkCudaError(hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost), "Failed to copy data for c");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}
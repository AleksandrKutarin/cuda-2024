#include "hip/hip_runtime.h"
#include "gelu_cuda.h"
#include <cmath>
#include <hip/hip_runtime.h>

__global__ void GeluKernel(const float* input, float* output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        output[idx] = 0.5f * x * (1.0f + tanh(sqrt(2.0f / M_PI) * (x + 0.044715f * x * x * x)));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    size_t inputCount = input.size();
    std::vector<float> output(inputCount);

    float* deviceInput;
    float* deviceOutput;
    hipMalloc(&deviceInput, inputCount * sizeof(float));
    hipMalloc(&deviceOutput, inputCount * sizeof(float));

    hipMemcpy(deviceInput, input.data(), inputCount * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (inputCount + blockSize - 1) / blockSize;

    GeluKernel<<<numBlocks, blockSize>>>(deviceInput, deviceOutput, inputCount);

    hipMemcpy(output.data(), deviceOutput, inputCount * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceInput);
    hipFree(deviceOutput);

    return output;
}
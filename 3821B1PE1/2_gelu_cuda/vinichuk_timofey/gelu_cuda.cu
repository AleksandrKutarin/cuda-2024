#include "hip/hip_runtime.h"
// Copyright (c) 2024 Vinichuk Timofey

#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "gelu_cuda.h"

__global__ void geluKernel(const float* input, float* output, size_t count) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < count) {
        constexpr float geluCoef1 = 0.044715f;
        constexpr float geluCoef2 = 0.7978845608f;
        float x = input[i];
        output[i] = 0.5f * x * (1.0f + tanhf(geluCoef2 * x * (1.0f + geluCoef1 * x * x)));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // �������� �������� device

    if (input.empty()) return {};

    auto size = input.size();

    size_t countBytes = size * sizeof(float);
    std::vector<float> output(size);



    float* input_block = nullptr;
    float* output_block = nullptr;

    hipMalloc(&input_block, size * sizeof(float));
    hipMalloc(&output_block, size * sizeof(float));
    hipMemcpy(input_block, input.data(), size * sizeof(float), hipMemcpyHostToDevice);
    auto blockSize = deviceProp.maxThreadsPerBlock;
    auto numBlocks = (size + blockSize - 1) / blockSize;

    geluKernel << <numBlocks, blockSize >> > (input_block, output_block, size);
    hipMemcpy(output.data(), output_block, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(input_block);
    hipFree(output_block);

    return output;
}

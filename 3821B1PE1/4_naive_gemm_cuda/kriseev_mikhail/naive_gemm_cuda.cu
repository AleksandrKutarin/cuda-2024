#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <vector>

__global__ void NaiveGemmKernel(const float *a, const float *b, float *output,
                                  int n) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n && j < n) {
    float res = 0.0f;
    for (int k = 0; k < n; k++) {
        res = fma(a[i * n + k], b[k * n + j], res);
    }
    output[i * n + j] = res;
  }
  
}

std::vector<float> NaiveGemmCUDA(const std::vector<float> &a,
                                 const std::vector<float> &b, int n) {
  std::vector<float> output(n * n);

  float *a_dev, *b_dev, *output_dev;

  hipMalloc(&a_dev, a.size() * sizeof(float));
  hipMalloc(&b_dev, b.size() * sizeof(float));
  hipMalloc(&output_dev, n * n * sizeof(float));

  hipMemcpy(a_dev, a.data(), a.size() * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_dev, b.data(), b.size() * sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 32;
  dim3 blockDim(blockSize, blockSize);
  dim3 gridDim((n + blockSize - 1) / blockSize, (n + blockSize - 1) / blockSize);

  NaiveGemmKernel<<<gridDim, blockDim>>>(a_dev, b_dev, output_dev, n);

  hipMemcpy(output.data(), output_dev, n * n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(a_dev);
  hipFree(b_dev);
  hipFree(output_dev);
  return output;
}
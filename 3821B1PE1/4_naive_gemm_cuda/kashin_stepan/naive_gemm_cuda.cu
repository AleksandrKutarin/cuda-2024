#include "hip/hip_runtime.h"
// Copyright (c) 2024 Kashin Stepan

#include <cstdlib>
#include <iostream>
#include <chrono>
#include <thread>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "naive_gemm_cuda.h"

#define BLOCK_DIM 32

__global__ void MatrixMultiplyKernel(const float* matrixA, const float* matrixB, float* matrixC,
                                     const size_t matrixSize)
{
    constexpr auto blockDim = BLOCK_DIM;
    __shared__ float sharedA[blockDim][blockDim];
    __shared__ float sharedB[blockDim][blockDim];

    size_t rowIdx = blockIdx.y * blockDim + threadIdx.y;
    size_t colIdx = blockIdx.x * blockDim + threadIdx.x;

    float tempResult = 0.0f;

    for (size_t k = 0; k < matrixSize; k += blockDim) {

        if (colIdx < matrixSize && (threadIdx.y + k) < matrixSize) {
            sharedB[threadIdx.y][threadIdx.x] = __ldg(&matrixB[(threadIdx.y + k) * matrixSize + colIdx]);
        } else {
            sharedB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (rowIdx < matrixSize && (threadIdx.x + k) < matrixSize) {
            sharedA[threadIdx.y][threadIdx.x] = __ldg(&matrixA[rowIdx * matrixSize + threadIdx.x + k]);
        } else {
            sharedA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (size_t l = 0; l < blockDim; ++l) {
            tempResult += sharedA[threadIdx.y][l] * sharedB[l][threadIdx.x];
        }

        __syncthreads();
    }

    if (rowIdx < matrixSize && colIdx < matrixSize) {
        matrixC[rowIdx * matrixSize + colIdx] = tempResult;
    }
}

std::vector<float> PerformMatrixMultiplicationCUDA(const std::vector<float>& matrixA,
                                                  const std::vector<float>& matrixB, int matrixSize) {
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);

    auto totalElements = matrixSize * matrixSize;
    std::vector<float> result(totalElements);
    auto totalSizeInBytes = totalElements * sizeof(float);

    dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);
    auto blocksPerGrid = (matrixSize + BLOCK_DIM - 1) / BLOCK_DIM;
    dim3 numBlocks(blocksPerGrid, blocksPerGrid);

    float *aDevice = nullptr;
    hipMalloc(&aDevice, totalSizeInBytes);

    float *bDevice = nullptr;
    hipMalloc(&bDevice, totalSizeInBytes);

    float *cDevice = nullptr;
    hipMalloc(&cDevice, totalSizeInBytes);

    hipMemcpy(aDevice, matrixA.data(), totalSizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(bDevice, matrixB.data(), totalSizeInBytes, hipMemcpyHostToDevice);

    std::this_thread::sleep_for(std::chrono::milliseconds(100));

    MatrixMultiplyKernel<<<numBlocks, threadsPerBlock>>>(aDevice, bDevice, cDevice, matrixSize);

    hipDeviceSynchronize();
    hipMemcpy(result.data(), cDevice, totalSizeInBytes, hipMemcpyDeviceToHost);

    hipFree(cDevice);
    hipFree(bDevice);
    hipFree(aDevice);

    return result;
}

#include "gemm_cublas.h"

#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float> &a,
                              const std::vector<float> &b, int n) {
  std::vector<float> output(n * n);

  float *a_dev;
  float *b_dev;
  float *output_dev;

  hipMalloc(&a_dev, a.size() * sizeof(float));
  hipMalloc(&b_dev, b.size() * sizeof(float));
  hipMalloc(&output_dev, output.size() * sizeof(float));

  hipMemcpy(a_dev, a.data(), a.size() * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_dev, b.data(), b.size() * sizeof(float), hipMemcpyHostToDevice);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float alpha = 1.0f;
  float beta = 0.0f;
  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n,n,n, &alpha, b_dev, n, a_dev, n, &beta, output_dev, n);

  hipMemcpy(output.data(), output_dev, output.size() * sizeof(float),
             hipMemcpyDeviceToHost);

  hipblasDestroy(handle);
  hipFree(a_dev);
  hipFree(b_dev);
  hipFree(output_dev);

  return output;
}

#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>

#include <iostream>

__global__ void normalize_kernel(float *data, int size, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    data[i] = data[i] / n;
  }
}

std::vector<float> FffCUFFT(const std::vector<float> &input, int batch) {
  std::vector<float> output(input.size());

  int n = input.size() / (batch * 2);

  hipfftHandle plan;
  hipfftComplex *data;

  hipMalloc((void **)&data, n * batch * sizeof(hipfftComplex));
  hipMemcpy(data, input.data(), input.size() * sizeof(float),
             hipMemcpyHostToDevice);

  hipfftPlanMany(&plan, 1, &n, NULL, 0, 0, NULL, 0, 0, HIPFFT_C2C, batch);

  hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

  hipDeviceSynchronize();

  hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);

  hipDeviceSynchronize();

  hipfftDestroy(plan);

  int blockSize = 64;
  int numBlocks = (output.size() + blockSize - 1) / blockSize;

  normalize_kernel<<<numBlocks, blockSize>>>(reinterpret_cast<float *>(data),
                                             output.size(), n);

  hipMemcpy(output.data(), data, output.size() * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(data);

  return output;
}

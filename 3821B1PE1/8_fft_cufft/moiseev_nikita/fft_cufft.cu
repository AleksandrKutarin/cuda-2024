#include "hip/hip_runtime.h"
// Copyright (c) 2024 Moiseev Nikita
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

__global__ void normalizeKernel(float* output_data, int total_size, float normalization_factor) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < total_size) {
        output_data[index] *= normalization_factor;
    }
}

std::vector<float> FFTCUFFT(const std::vector<float>& input_data, int batch_size) {
    const int total_size = input_data.size();
    std::vector<float> normalized_output(total_size);
    int batch_elements = (total_size / batch_size) >> 1;

    int data_size_bytes = sizeof(hipfftComplex) * batch_elements * batch_size;
    hipfftHandle fft_plan;
    hipfftPlan1d(&fft_plan, batch_elements, HIPFFT_C2C, batch_size);
    hipfftComplex* device_data;

    hipMalloc(&device_data, data_size_bytes);
    hipMemcpy(device_data, input_data.data(), data_size_bytes, hipMemcpyHostToDevice);
    hipfftExecC2C(fft_plan, device_data, device_data, HIPFFT_FORWARD);
    hipfftExecC2C(fft_plan, device_data, device_data, HIPFFT_BACKWARD);

    hipDeviceProp_t device_properties;
    hipGetDeviceProperties(&device_properties, 0);

    size_t threads_per_block = device_properties.maxThreadsPerBlock;
    size_t blocks_per_grid = (total_size + threads_per_block - 1) / threads_per_block;
    float normalization_factor = 1.0f / static_cast<float>(batch_elements);

    normalizeKernel<<<blocks_per_grid, threads_per_block>>>(
        reinterpret_cast<float*>(device_data), total_size, normalization_factor);

    hipMemcpy(normalized_output.data(), device_data, data_size_bytes, hipMemcpyDeviceToHost);
    hipfftDestroy(fft_plan);
    hipFree(device_data);

    return normalized_output;
}

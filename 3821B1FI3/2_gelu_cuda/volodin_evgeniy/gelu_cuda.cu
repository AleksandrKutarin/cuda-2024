#include "hip/hip_runtime.h"
// Copyright (c) 2024 Volodin Evgeniy
#include "gelu_cuda.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <cmath>
#include <algorithm>

__global__ void gelu_kernel(const float* input, float* output, std::size_t size) {
    const float sqrt_2pi = sqrtf(2.0f / M_PI);
    const float coeff_cubic = 0.044715f;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        output[idx] = 0.5f * x * (1.0f + tanh(sqrt_2pi * (x + coeff_cubic * x * x * x)));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    if (input.empty()) {
        throw std::invalid_argument("Input vector is empty!");
    }

    hipDeviceProp_t deviceProp;
    hipError_t cudaErr = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaErr != hipSuccess) {
        throw std::runtime_error("Failed to get device properties!");
    }

    std::size_t size = input.size();
    std::vector<float> output(size);

    float* ptr_input = nullptr;
    float* ptr_output = nullptr;


    cudaErr = hipMalloc(&ptr_input, size * sizeof(float));
    if (cudaErr != hipSuccess) {
        throw std::runtime_error("Failed to allocate device memory for input.");
    }

    cudaErr = hipMalloc(&ptr_output, size * sizeof(float));
    if (cudaErr != hipSuccess) {
        throw std::runtime_error("Failed to allocate device memory for output.");
    }

    cudaErr = hipMemcpy(ptr_input, input.data(), size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaErr != hipSuccess) {
        hipFree(ptr_input);
        hipFree(ptr_output);
        throw std::runtime_error("Failed to copy input data to device.");
    }

    int blockSize = std::min(deviceProp.maxThreadsPerBlock, 256);
    int numBlocks = (size + blockSize - 1) / blockSize;

    gelu_kernel<<<numBlocks, blockSize>>>(ptr_input, ptr_output, size);

    cudaErr = hipDeviceSynchronize();
    if (cudaErr != hipSuccess) {
        hipFree(ptr_input);
        hipFree(ptr_output);
        throw std::runtime_error("CUDA kernel execution failed.");
    }

    cudaErr = hipMemcpy(output.data(), ptr_output, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaErr != hipSuccess) {
        hipFree(ptr_input);
        hipFree(ptr_output);
        throw std::runtime_error("Failed to copy output data to host.");
    }

    hipFree(ptr_input);
    hipFree(ptr_output);

    return output;
}

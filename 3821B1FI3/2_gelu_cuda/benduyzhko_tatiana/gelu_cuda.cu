#include <vector>
#include <cstdlib>

#include <hip/hip_runtime.h>



const float sqrt2pi = 0.797884f;

__global__ void kernel(const float* sample, float* result,
                            size_t elemCount) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;

  if (id < elemCount) {
    const float num = sample[id];
    result[id] = 0.5f * num *
                 (1.0f + tanhf(sqrt2pi * num * (1.0f + 0.044715f * num * num)));
  }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
  const size_t size = input.size();
  std::vector<float> output(size);

  size_t sizeInBytes = size * sizeof(*input.data());

  float* d_input;
  float* d_output;
  hipMalloc(&d_input, sizeInBytes);
  hipMalloc(&d_output, sizeInBytes);

  hipMemcpy(d_input, input.data(), sizeInBytes, hipMemcpyHostToDevice);

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
  size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);

  hipMemcpy(output.data(), d_output, sizeInBytes, hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
  return output;
}

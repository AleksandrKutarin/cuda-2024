#include <vector>
#include <cstdlib>

#include <hip/hip_runtime.h>


__global__ void myKernel(const float* a, const float* b, float* const c,
                         const size_t size) {
  size_t mIdx = blockIdx.y * blockDim.y + threadIdx.y;
  size_t nIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (mIdx < size && nIdx < size) {
    float cVal = 0.0f;
    for (size_t k = 0; k < size; ++k)
      cVal += a[mIdx * size + k] * b[size * k + nIdx];
    c[mIdx * size + nIdx] = cVal;
  }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b, int n) {
  std::vector<float> c(n * n);

  size_t sizeInBytes = n * n * sizeof(*a.data());

  float* device_a;
  float* device_b;
  float* device_c;
  hipMalloc(&device_a, sizeInBytes);
  hipMalloc(&device_b, sizeInBytes);
  hipMalloc(&device_c, sizeInBytes);

  hipMemcpy(device_a, a.data(), sizeInBytes, hipMemcpyHostToDevice);
  hipMemcpy(device_b, b.data(), sizeInBytes, hipMemcpyHostToDevice);

  const size_t sizeAxis = 32u;
  dim3 threadsPerBlock(sizeAxis, sizeAxis);
  dim3 numBlocks((n + sizeAxis - 1) / sizeAxis,
                 (n + sizeAxis - 1) / sizeAxis);

  myKernel<<<numBlocks, threadsPerBlock>>>(device_a, device_b, device_c, n);

  hipMemcpy(c.data(), device_c, sizeInBytes, hipMemcpyDeviceToHost);

  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);
  return c;
}

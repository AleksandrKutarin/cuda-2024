#include "hip/hip_runtime.h"
#include <cmath>

#include "naive_gemm_cuda.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void naiveGemmKernel(const float* a, const float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n) {
        float sum = 0.0f;
        for (int r = 0; r < n; ++r) {
            sum += a[i * n + r] * b[r * n + j];
        }
        c[i * n + j] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    
    std::vector<float> c(n * n, 0.0f);

    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, n * n * sizeof(float));
    hipMalloc(&d_b, n * n * sizeof(float));
    hipMalloc(&d_c, n * n * sizeof(float));

    hipMemcpy(d_a, a.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), n * n * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    naiveGemmKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    hipMemcpy(c.data(), d_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}
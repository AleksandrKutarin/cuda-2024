#include "hip/hip_runtime.h"
// Copyright (c) 2024 Volodin Evgeniy
#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>
#include <iostream>

__global__ void gemm_kernel(const float* a, const float* b, float* c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a, const std::vector<float>& b, int n) {
    if (a.size() != n * n || b.size() != n * n) {
        throw std::invalid_argument("Matrix size does not match the specified n*n dimensions!");
    }

    std::vector<float> c(n * n, 0.0f);

    float *ptr_a, *ptr_b, *ptr_c;

    hipError_t cudaErr = hipMalloc(&ptr_a, n * n * sizeof(float));
    if (cudaErr != hipSuccess) {
        throw std::runtime_error("Failed to allocate device memory for matrix A.");
    }

    cudaErr = hipMalloc(&ptr_b, n * n * sizeof(float));
    if (cudaErr != hipSuccess) {
        throw std::runtime_error("Failed to allocate device memory for matrix B.");
    }

    cudaErr = hipMalloc(&ptr_c, n * n * sizeof(float));
    if (cudaErr != hipSuccess) {
        hipFree(ptr_a);
        hipFree(ptr_b);
        throw std::runtime_error("Failed to allocate device memory for matrix C.");
    }

    cudaErr = hipMemcpy(ptr_a, a.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    if (cudaErr != hipSuccess) {
        hipFree(ptr_a);
        hipFree(ptr_b);
        hipFree(ptr_c);
        throw std::runtime_error("Failed to copy matrix A to device.");
    }

    cudaErr = hipMemcpy(ptr_b, b.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    if (cudaErr != hipSuccess) {
        hipFree(ptr_a);
        hipFree(ptr_b);
        hipFree(ptr_c);
        throw std::runtime_error("Failed to copy matrix B to device.");
    }

    int blockSize = 32;

    dim3 block(blockSize, blockSize);
    dim3 grid((n + block.x - 1) / block.x, (n + block.y - 1) / block.y);
    
    gemm_kernel<<<grid, block>>>(ptr_a, ptr_b, ptr_c, n);

    hipDeviceSynchronize();

    cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        hipFree(ptr_a);
        hipFree(ptr_b);
        hipFree(ptr_c);
        throw std::runtime_error("Kernel launch failed.");
    }

    cudaErr = hipMemcpy(c.data(), ptr_c, n * n * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaErr != hipSuccess) {
        hipFree(ptr_a);
        hipFree(ptr_b);
        hipFree(ptr_c);
        throw std::runtime_error("Failed to copy result matrix C to host.");
    }

    hipFree(ptr_a);
    hipFree(ptr_b);
    hipFree(ptr_c);

    return c;
}
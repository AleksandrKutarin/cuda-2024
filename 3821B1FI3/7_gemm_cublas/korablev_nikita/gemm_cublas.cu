// Copyright (c) 2024 Korablev Nikita
#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    std::vector<float> c(n * n, 0.0f);

    size_t size = n * n * sizeof(float);
    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 n, n, n, &alpha,
                 d_b, HIP_R_32F, n,
                 d_a, HIP_R_32F, n,
                 &beta, d_c, HIP_R_32F, n,
                 HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);
    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipblasDestroy(handle);
    return c;
}

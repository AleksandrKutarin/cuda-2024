// Copyright (c) 2024 Ivanov Nikita
#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    std::vector<float> c(n * n, 0.0f);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float* d_a;
    float* d_b;
    float* d_c;
    size_t sizeInBytes = n * n * sizeof(float);
    hipMalloc(&d_a, sizeInBytes);
    hipMalloc(&d_b, sizeInBytes);
    hipMalloc(&d_c, sizeInBytes);

    hipMemcpy(d_a, a.data(), sizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), sizeInBytes, hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);

    hipblasGemmEx(handle,
                 HIPBLAS_OP_N, HIPBLAS_OP_N,
                 n, n, n,
                 &alpha,
                 d_b, HIP_R_32F, n,
                 d_a, HIP_R_32F, n,
                 &beta,
                 d_c, HIP_R_32F, n,
                 HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);

    hipMemcpy(c.data(), d_c, sizeInBytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}

#include "gemm_cublas.h"

void CheckCudaStatus(hipError_t status)
{
    if (status != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime API error %d: %s\n", status, hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}

void CheckCublasStatus(hipblasStatus_t status)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuBLAS API error %d\n", status);
        exit(EXIT_FAILURE);
    }
}

std::vector<float> GemmCUBLAS(const std::vector<float> &a,
                              const std::vector<float> &b,
                              int n)
{
    assert(a.size() == n * n && b.size() == n * n && "Matrix size mismatch");

    float *d_A, *d_B, *d_C;
    CheckCudaStatus(hipMalloc((void **)&d_A, n * n * sizeof(float)));
    CheckCudaStatus(hipMalloc((void **)&d_B, n * n * sizeof(float)));
    CheckCudaStatus(hipMalloc((void **)&d_C, n * n * sizeof(float)));

    CheckCudaStatus(hipMemcpy(d_A, a.data(), n * n * sizeof(float), hipMemcpyHostToDevice));
    CheckCudaStatus(hipMemcpy(d_B, b.data(), n * n * sizeof(float), hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CheckCublasStatus(hipblasCreate(&handle));

    const float alpha = 1.0f;
    const float beta = 0.0f;
    CheckCublasStatus(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                  n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n));

    std::vector<float> h_C(n * n);

    CheckCudaStatus(hipMemcpy(h_C.data(), d_C, n * n * sizeof(float), hipMemcpyDeviceToHost));

    CheckCudaStatus(hipFree(d_A));
    CheckCudaStatus(hipFree(d_B));
    CheckCudaStatus(hipFree(d_C));
    CheckCublasStatus(hipblasDestroy(handle));

    return h_C;
}

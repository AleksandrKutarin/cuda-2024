#include "hip/hip_runtime.h"
#include "gelu_cuda.h"

// CUDA Kernel for GELU computation
__global__ void geluKernel(float *input, float *output, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        float x = input[idx];
        float gelu = 0.5f * x * (1 + tanh(sqrtf(2 / 3.14159f) * (x + 0.044715f * x * x)));
        output[idx] = gelu;
    }
}

std::vector<float> GeluCUDA(const std::vector<float> &input)
{
    int size = input.size();
    if (size == 0)
        return {}; // edge case: empty input

    // Allocate device memory
    float *d_input;
    float *d_output;
    hipMalloc((void **)&d_input, size * sizeof(float));
    hipMalloc((void **)&d_output, size * sizeof(float));

    // Copy input from host to device
    hipMemcpy(d_input, input.data(), size * sizeof(float), hipMemcpyHostToDevice);

    // Launch CUDA kernel
    int blockSize = 32;                                 // adjust based on your GPU's capabilities
    int numBlocks = (size + blockSize - 1) / blockSize; // ceiling division
    geluKernel<<<numBlocks, blockSize>>>(d_input, d_output, size);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    // Copy output from device to host
    std::vector<float> output(size);
    hipMemcpy(output.data(), d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_input);
    hipFree(d_output);

    return output;
}
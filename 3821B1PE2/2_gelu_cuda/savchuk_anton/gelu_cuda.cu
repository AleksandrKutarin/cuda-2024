#include "hip/hip_runtime.h"
﻿#include "gelu_cuda.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <stdexcept>

#ifndef M_PI
#define M_PI 3.14159265358979323846 // Определение числа π
#endif

__global__ void GeluKernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        output[idx] = 0.5f * x * (1.0f + tanhf(sqrtf(2.0f / M_PI) * (x + 0.044715f * x * x * x)));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    int size = input.size();

    if (size == 0) {
        return {};
    }

    float* d_input;
    float* d_output;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    hipMemcpy(d_input, input.data(), size * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    GeluKernel << <blocksPerGrid, threadsPerBlock >> > (d_input, d_output, size);

    std::vector<float> output(size);
    hipMemcpy(output.data(), d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}

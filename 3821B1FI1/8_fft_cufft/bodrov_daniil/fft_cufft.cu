#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>

// Макрос для проверки ошибок CUDA
#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        throw std::runtime_error(std::string("CUDA Error: ") + hipGetErrorString(err)); \
    }

// Макрос для проверки ошибок cuFFT
#define CUFFT_CHECK(err) \
    if (err != HIPFFT_SUCCESS) { \
        throw std::runtime_error(std::string("cuFFT Error: ") + cufftGetErrorString(err)); \
    }

// Функция для получения строкового описания ошибок cuFFT
const char* cufftGetErrorString(hipfftResult error) {
    switch(error) {
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED:
            return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE:
            return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE:
            return "HIPFFT_INVALID_SIZE";
        default:
            return "Unknown cuFFT error";
    }
}

// CUDA-ядро для нормализации данных на устройстве
__global__ void normalize(hipfftComplex* data, int size, float norm_factor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx].x *= norm_factor;
        data[idx].y *= norm_factor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    // Проверка корректности размера входных данных
    if (input.size() % (2 * batch) != 0) {
        throw std::invalid_argument("Размер входного массива не соответствует формату (real, imaginary) для заданного batch.");
    }

    // Вычисление размера одного сигнала
    int n = input.size() / (2 * batch);

    // Общее количество комплексных чисел
    int total_elements = n * batch;

    // Размер данных в байтах
    size_t bytes = sizeof(hipfftComplex) * total_elements;

    // Указатель на данные на устройстве
    hipfftComplex* d_data = nullptr;

    // Выделение памяти на устройстве
    CUDA_CHECK(hipMalloc((void**)&d_data, bytes));

    // Копирование данных с хоста на устройство
    CUDA_CHECK(hipMemcpy(d_data, input.data(), bytes, hipMemcpyHostToDevice));

    // Создание дескриптора cuFFT
    hipfftHandle plan;

    // Создание плана для FFT
    CUFFT_CHECK(hipfftPlan1d(&plan, n, HIPFFT_C2C, batch));

    // Выполнение прямого FFT (in-place)
    CUFFT_CHECK(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));

    // Выполнение обратного FFT (in-place)
    CUFFT_CHECK(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD));

    // Нормализация результата на устройстве
    int threadsPerBlock = 256;
    int blocksPerGrid = (total_elements + threadsPerBlock - 1) / threadsPerBlock;
    float norm_factor = 1.0f / static_cast<float>(n);

    normalize<<<blocksPerGrid, threadsPerBlock>>>(d_data, total_elements, norm_factor);

    // Проверка на ошибки после запуска ядра
    CUDA_CHECK(hipGetLastError());

    // Синхронизация устройства
    CUDA_CHECK(hipDeviceSynchronize());

    // Выделение памяти для результата на хосте
    std::vector<float> output(2 * total_elements);

    // Копирование результата с устройства на хост
    CUDA_CHECK(hipMemcpy(output.data(), d_data, bytes, hipMemcpyDeviceToHost));

    // Освобождение плана
    CUFFT_CHECK(hipfftDestroy(plan));

    // Освобождение памяти на устройстве
    CUDA_CHECK(hipFree(d_data));

    return output;
}
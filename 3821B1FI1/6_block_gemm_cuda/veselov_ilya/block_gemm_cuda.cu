#include "hip/hip_runtime.h"
#include "block_gemm_cuda.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_SIZE 32

__global__ void blockGemmKernel(const float* A, const float* B, float* C, int n) {
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int row = threadIdx.y;
    int col = threadIdx.x;

    int cIndex = (blockRow * BLOCK_SIZE + row) * n + (blockCol * BLOCK_SIZE + col);
    float cValue = 0.0f;

    __shared__ float sharedA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sharedB[BLOCK_SIZE][BLOCK_SIZE];

    int numBlocks = n / BLOCK_SIZE;

    for (int m = 0; m < numBlocks; ++m) {
        sharedA[row][col] = A[(blockRow * BLOCK_SIZE + row) * n + (m * BLOCK_SIZE + col)];
        sharedB[row][col] = B[(m * BLOCK_SIZE + row) * n + (blockCol * BLOCK_SIZE + col)];

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            cValue += sharedA[row][k] * sharedB[k][col];
        }

        __syncthreads();
    }

    C[cIndex] = cValue;
}

std::vector<float> BlockGemmCUDA(const std::vector<float>& a, const std::vector<float>& b, int n) {
    size_t matrixSize = n * n * sizeof(float);
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, matrixSize);
    hipMalloc((void**)&d_B, matrixSize);
    hipMalloc((void**)&d_C, matrixSize);

    hipMemcpy(d_A, a.data(), matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b.data(), matrixSize, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(n / BLOCK_SIZE, n / BLOCK_SIZE);

    blockGemmKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, n);

    std::vector<float> c(n * n);
    hipMemcpy(c.data(), d_C, matrixSize, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return c;
}

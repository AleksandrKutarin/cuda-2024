#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdexcept>
#include <iostream>

// Класс для обработки ошибок CUDA и cuBLAS
class CUBLASError : public std::runtime_error {
public:
    explicit CUBLASError(const char* message)
        : std::runtime_error(message) {}
};

// Макрос для проверки ошибок CUDA
#define CHECK_CUDA(call)                                              \
    do {                                                             \
        hipError_t err = (call);                                    \
        if (err != hipSuccess) {                                    \
            throw CUBLASError(hipGetErrorString(err));              \
        }                                                            \
    } while (0)

// Макрос для проверки ошибок cuBLAS
#define CHECK_CUBLAS(call)                                            \
    do {                                                             \
        hipblasStatus_t status = (call);                              \
        if (status != HIPBLAS_STATUS_SUCCESS) {                       \
            throw CUBLASError("cuBLAS operation failed");            \
        }                                                            \
    } while (0)

std::vector<float> GemmCUBLAS(const std::vector<float>& matrixA,
                                        const std::vector<float>& matrixB,
                                        int size) {
    // Результирующий вектор
    std::vector<float> matrixC(size * size);

    // Создание дескриптора cuBLAS
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Создание потока CUDA для асинхронных операций
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));
    CHECK_CUBLAS(hipblasSetStream(handle, stream));

    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;
    size_t bytes = size * size * sizeof(float);

    try {
        // Выделение памяти на устройстве
        CHECK_CUDA(hipMalloc(&d_A, bytes));
        CHECK_CUDA(hipMalloc(&d_B, bytes));
        CHECK_CUDA(hipMalloc(&d_C, bytes));

        // Копирование данных с хоста на устройство (асинхронно)
        CHECK_CUBLAS(hipblasSetMatrix(size, size, sizeof(float),
                                     matrixA.data(), size,
                                     d_A, size));
        CHECK_CUBLAS(hipblasSetMatrix(size, size, sizeof(float),
                                     matrixB.data(), size,
                                     d_B, size));

        // Параметры умножения матриц
        const float alpha = 1.0f;
        const float beta = 0.0f;

        // Выполнение умножения матриц: C = A * B
        CHECK_CUBLAS(hipblasSgemm(handle,
                                 HIPBLAS_OP_N, HIPBLAS_OP_N,
                                 size, size, size,
                                 &alpha,
                                 d_B, size,
                                 d_A, size,
                                 &beta,
                                 d_C, size));

        // Копирование результата с устройства на хост (асинхронно)
        CHECK_CUBLAS(hipblasGetMatrix(size, size, sizeof(float),
                                     d_C, size,
                                     matrixC.data(), size));

        // Синхронизация потока для завершения всех операций
        CHECK_CUDA(hipStreamSynchronize(stream));
    }
    catch (...) {
        // Освобождение ресурсов в случае исключения
        if (d_A) hipFree(d_A);
        if (d_B) hipFree(d_B);
        if (d_C) hipFree(d_C);
        hipStreamDestroy(stream);
        hipblasDestroy(handle);
        throw; // Переброс исключения
    }

    // Освобождение ресурсов
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipStreamDestroy(stream);
    hipblasDestroy(handle);

    return matrixC;
}